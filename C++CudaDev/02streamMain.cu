#include <hip/hip_runtime.h>
#include <iostream>

// CUDA核函数定义
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // 分配主机内存
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // 初始化主机数据
    for(int i = 0; i < numElements; ++i)
    {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // 分配设备内存
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // cudaMalloc((void **)&d_A, size);
    // cudaMalloc((void **)&d_B, size);
    // cudaMalloc((void **)&d_C, size);

    // 创建两个流
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // 复制数据到设备，并在两个不同的流中执行向量加法
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream1);
    vectorAdd<<<(numElements + 255) / 256, 256, 0, stream1>>>(d_A, d_B, d_C, numElements);
    
    // 在第二个流中再次执行相同的操作
    // 为了示例简单，这里再次使用同样的输入数据
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream2);
    vectorAdd<<<(numElements + 255) / 256, 256, 0, stream2>>>(d_A, d_B, d_C, numElements);

    // 等待流完成
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // 复制结果回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 检查错误并打印几个结果
    for (int i = 0; i < 5; ++i)
    {
        std::cout << "C[" << i << "] = " << h_C[i] << "\n";
    }
    
    // 清理
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
